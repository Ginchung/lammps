#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 AdvanceSoft Corporation
 *
 * This software is released under the MIT License.
 * http://opensource.org/licenses/mit-license.php
 */

#include "nnp_symm_func_gpu.h"

SymmFuncGPU::SymmFuncGPU(int numElems, bool tanhCutFunc, bool elemWeight, int sizeRad, int sizeAng,
                         nnpreal rcutRad, nnpreal rcutAng, int cutoffMode) : SymmFunc(numElems, tanhCutFunc, elemWeight)
{
    if (sizeRad < 1)
    {
        stop_by_error("size of radius basis is not positive.");
    }

    if (sizeAng < 0)
    {
        stop_by_error("size of angle basis is negative.");
    }

    if (rcutRad <= ZERO)
    {
        stop_by_error("cutoff for radius is not positive.");
    }

    if (sizeAng > 0 && rcutAng <= ZERO)
    {
        stop_by_error("cutoff for angle is not positive.");
    }

    this->maxThreadsPerBlock = 1;

    this->sizeRad = sizeRad;
    this->sizeAng = sizeAng;

    if (this->elemWeight)
    {
        this->numRadBasis = this->sizeRad;
        this->numAngBasis = this->sizeAng;
    }
    else
    {
        this->numRadBasis = this->sizeRad * this->numElems;
        this->numAngBasis = this->sizeAng * (this->numElems * (this->numElems + 1) / 2);
    }

    this->numBasis = this->numRadBasis + this->numAngBasis;

    this->rcutRad = rcutRad;
    this->rcutAng = rcutAng;

    this->sizeLenAtoms = 0;
    this->sizeTotNeigh = 0;

    if (cutoffMode == CUTOFF_MODE_SINGLE)
    {
        this->sizePosNeighbor = 6;
    }
    else if (cutoffMode == CUTOFF_MODE_DOUBLE || cutoffMode == CUTOFF_MODE_IPSO)
    {
        this->sizePosNeighbor = 8;
    }
    else
    {
        this->sizePosNeighbor = 4;
    }

    this->numNeighs        = nullptr;
    this->numNeighs_d      = nullptr;
    this->idxNeighs        = nullptr;
    this->idxNeighs_d      = nullptr;
    this->elementAll       = nullptr;
    this->elementAll_d     = nullptr;
    this->posNeighborAll   = nullptr;
    this->posNeighborAll_d = nullptr;
    this->symmDataAll      = nullptr;
    this->symmDataAll_d    = nullptr;
    this->symmDiffAll      = nullptr;
    this->symmDiffAll_d    = nullptr;

#ifdef _NNP_SINGLE
    hipError_t error = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
#else
    hipError_t error = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
#endif

    if (error != hipSuccess)
    {
        char message[512];
        sprintf(message, "error of hipDeviceSetSharedMemConfig: %s\n", hipGetErrorString(error));
        stop_by_error(message);
    }
}

SymmFuncGPU::~SymmFuncGPU()
{
    if (this->numNeighs        != nullptr) hipHostFree(this->numNeighs);
    if (this->numNeighs_d      != nullptr) hipFree    (this->numNeighs_d);
    if (this->idxNeighs        != nullptr) hipHostFree(this->idxNeighs);
    if (this->idxNeighs_d      != nullptr) hipFree    (this->idxNeighs_d);
    if (this->elementAll       != nullptr) hipHostFree(this->elementAll);
    if (this->elementAll_d     != nullptr) hipFree    (this->elementAll_d);
    if (this->posNeighborAll   != nullptr) hipHostFree(this->posNeighborAll);
    if (this->posNeighborAll_d != nullptr) hipFree    (this->posNeighborAll_d);
    if (this->symmDataAll      != nullptr) hipHostFree(this->symmDataAll);
    if (this->symmDataAll_d    != nullptr) hipFree    (this->symmDataAll_d);
    if (this->symmDiffAll      != nullptr) hipHostFree(this->symmDiffAll);
    if (this->symmDiffAll_d    != nullptr) hipFree    (this->symmDiffAll_d);
}

void SymmFuncGPU::calculate(int lenAtoms, int* numNeighbor, int** elemNeighbor, nnpreal*** posNeighbor,
                            nnpreal** symmData, nnpreal** symmDiff);
{
    if (numAtoms < 0)
    {
        stop_by_error("#atoms is not positive.");
    }

    if (numNeighbor == nullptr || elemNeighbor == nullptr || posNeighbor == nullptr)
    {
        stop_by_error("neighbor is null.");
    }

    if (symmData == nullptr)
    {
        stop_by_error("symmData is null.");
    }

    if (symmDiff == nullptr)
    {
        stop_by_error("symmDiff is null.");
    }

    // define varialbes
    int iatom;
    int ineigh, jneigh;
    int ifree;
    int ibase;
    int ipos;

    int numNeigh;
    int idxNeigh;
    int maxNeigh;
    int totNeigh;

    int numPos;
    int idxPos;

    int idxBase;
    int idxDiff;

    int numModeBatchs;
    int modesPerBatch;
    int dimBasis;

    dim3 grid;
    dim3 block;

    size_t sizeShared;

    // allocate memory about lenAtoms
    if (this->sizeLenAtoms < lenAtoms)
    {
        if (this->numNeighs   != nullptr) hipHostFree(this->numNeighs);
        if (this->numNeighs_d != nullptr) hipFree    (this->numNeighs_d);
        if (this->idxNeighs   != nullptr) hipHostFree(this->idxNeighs);
        if (this->idxNeighs_d != nullptr) hipFree    (this->idxNeighs_d);

        hipHostMalloc(&(this->numNeighs),   sizeof(int) * lenAtoms);
        hipMalloc    (&(this->numNeighs_d), sizeof(int) * lenAtoms);
        hipHostMalloc(&(this->idxNeighs),   sizeof(int) * lenAtoms);
        hipMalloc    (&(this->idxNeighs_d), sizeof(int) * lenAtoms);

        this->sizeLenAtoms = lenAtoms;
    }

    // count neighbors
    maxNeigh = 0;
    totNeigh = 0;

    for (iatom = 0; iatom < lenAtoms; ++iatom)
    {
        numNeigh = numNeighbor[iatom];
        this->numNeighs[iatom] = numNeigh;
        this->idxNeighs[iatom] = totNeigh;

        maxNeigh  = max(maxNeigh, numNeigh);
        totNeigh += numNeigh;
    }

    if (maxNeigh > this->maxThreadsPerBlock)
    {
        stop_by_error("too less #threads a block for GPU.");
    }

    if (maxNeigh < 1 || totNeigh < 1)
    {
        #pragma omp parallel for private (iatom, ibase, ineigh, numNeigh, ifree)
        for (iatom = 0; iatom < lenAtoms; ++iatom)
        {
            for (ibase = 0; ibase < this->numBasis; ++ibase)
            {
                symmData[iatom][ibase] = ZERO;
            }

            for (ibase = 0; ibase < this->numBasis; ++ibase)
            {
                symmDiff[iatom][ibase + 0 * this->numBasis] = ZERO;
                symmDiff[iatom][ibase + 1 * this->numBasis] = ZERO;
                symmDiff[iatom][ibase + 2 * this->numBasis] = ZERO;
            }

            numNeigh = this->numNeighs[iatom];

            if (numNeigh < 1)
            {
                continue;
            }

            for (ineigh = 0; ineigh < numNeigh; ++ineigh)
            {
                ifree   = 3 * (ineigh + 1);

                for (ibase = 0; ibase < this->numBasis; ++ibase)
                {
                    symmDiff[iatom][ibase + (ifree + 0) * this->numBasis] = ZERO;
                    symmDiff[iatom][ibase + (ifree + 1) * this->numBasis] = ZERO;
                    symmDiff[iatom][ibase + (ifree + 2) * this->numBasis] = ZERO;
                }
            }
        }

        return;
    }

    // allocate memory about totNeigh
    if (this->sizeTotNeigh < totNeigh)
    {
        if (this->elementAll       != nullptr) hipHostFree(this->elementAll);
        if (this->elementAll_d     != nullptr) hipFree    (this->elementAll_d);
        if (this->posNeighborAll   != nullptr) hipHostFree(this->posNeighborAll);
        if (this->posNeighborAll_d != nullptr) hipFree    (this->posNeighborAll_d);
        if (this->symmDataAll      != nullptr) hipHostFree(this->symmDataAll);
        if (this->symmDataAll_d    != nullptr) hipFree    (this->symmDataAll_d);
        if (this->symmDiffAll      != nullptr) hipHostFree(this->symmDiffAll);
        if (this->symmDiffAll_d    != nullptr) hipFree    (this->symmDiffAll_d);

        hipHostMalloc(&(this->elementAll),       sizeof(gint)    * totNeigh);
        hipMalloc    (&(this->elementAll_d),     sizeof(gint)    * totNeigh);
        hipHostMalloc(&(this->posNeighborAll),   sizeof(nnpreal) * this->sizePosNeighbor * totNeigh);
        hipMalloc    (&(this->posNeighborAll_d), sizeof(nnpreal) * this->sizePosNeighbor * totNeigh);
        hipHostMalloc(&(this->symmDataAll),      sizeof(nnpreal) * this->numBasis * totNeigh);
        hipMalloc    (&(this->symmDataAll_d),    sizeof(nnpreal) * this->numBasis * totNeigh);
        hipHostMalloc(&(this->symmDiffAll),      sizeof(nnpreal) * this->numBasis * 3 * totNeigh);
        hipMalloc    (&(this->symmDiffAll_d),    sizeof(nnpreal) * this->numBasis * 3 * totNeigh);

        this->sizeTotNeigh = totNeigh;
    }

    // serialize all data of neighbors
    #pragma omp parallel for private (iatom, ineigh, jneigh, numNeigh, idxNeigh, ipos, numPos, idxPos)
    for (iatom = 0; iatom < lenAtoms; ++iatom)
    {
        numNeigh = this->numNeighs[iatom];
        idxNeigh = this->idxNeighs[iatom];

        if (numNeigh < 1 || idxNeigh < 0)
        {
            continue;
        }

        // element / atomnum of neighbor atoms
        for (ineigh = 0; ineigh < numNeigh; ++ineigh)
        {
            this->elementAll[ineigh + idxNeigh] = elemNeighbor[iatom][ineigh]
        }

        // positions of neighbor atoms
        for (ineigh = 0; ineigh < numNeigh; ++ineigh)
        {
            jneigh = ineigh + idxNeigh;
            numPos = this->sizePosNeighbor;
            idxPos = jneigh * numPos;

            for (ipos = 0; ipos < numPos; ++ipos)
            {
                this->posNeighborAll[ipos + idxPos] = posNeighbor[iatom][ineigh][ipos];
            }
        }
    }

    // copy memory host -> gpu
    hipMemcpy(this->numNeighs_d,      this->numNeighs,      sizeof(int)     * lenAtoms,                         hipMemcpyHostToDevice);
    hipMemcpy(this->idxNeighs_d,      this->idxNeighs,      sizeof(int)     * lenAtoms,                         hipMemcpyHostToDevice);
    hipMemcpy(this->elementAll_d,     this->elementAll,     sizeof(gint)    * totNeigh,                         hipMemcpyHostToDevice);
    hipMemcpy(this->posNeighborAll_d, this->posNeighborAll, sizeof(nnpreal) * totNeigh * this->sizePosNeighbor, hipMemcpyHostToDevice);

    /*
     * radial part
     *   imode   -> threadIdx.x + blockIdx.y * blockDim.x
     *   ineigh1 -> threadIdx.y
     *   iatom   -> blockIdx.x
     *   (num of mode batch) -> blockDim.x
     *   (idx of mode batch) -> blockIdx.y
     */
    if (this->sizeRad > 0)
    {
        this->getSizeOfModeBatchs(&numModeBatchs, &modesPerBatch, this->sizeRad, maxNeigh);

        block = dim3(modesPerBatch, maxNeigh, 1);
        grid  = dim3(lenAtoms, numModeBatchs, 1);

        this->calculateRadial(grid, block);
    }

    /*
     * angular  part
     *   imode   -> threadIdx.x + blockIdx.y * blockDim.x
     *   ineigh1 -> threadIdx.y
     *   iatom   -> blockIdx.x
     *   (num of mode batch) -> blockDim.x
     *   (idx of mode batch) -> blockIdx.y
     */
    if (this->sizeAng > 0)
    {
        this->getSizeOfModeBatchs(&numModeBatchs, &modesPerBatch, this->sizeAng, maxNeigh);

        block = dim3(modesPerBatch, maxNeigh, 1);
        grid  = dim3(lenAtoms, numModeBatchs, 1);

        sizeShared = sizeof(gint)    * maxNeigh
                   + sizeof(nnpreal) * maxNeigh * 5;

        if (this->elemWeight)
        {
            this->calculateAnglarElemWeight(grid, block, sizeShared);
        }
        else
        {
            dimBasis = this->numElems * (this->numElems + 1) / 2;

            if (dimBasis > MAX_ELEMENT_PAIRS)
            {
                stop_by_error("too much elements for symmetry functions on GPU, please use CPU.");
            }

            this->calculateAnglarNotElemWeight(grid, block, sizeShared, dimBasis);
        }
    }

    // copy memory gpu -> host
    hipMemcpy(this->symmDataAll, this->symmDataAll_d, sizeof(nnpreal) * this->numBasis *     totNeigh, hipMemcpyDeviceToHost);
    hipMemcpy(this->symmDiffAll, this->symmDiffAll_d, sizeof(nnpreal) * this->numBasis * 3 * totNeigh, hipMemcpyDeviceToHost);

    #pragma omp parallel for private (iatom, ineigh, jneigh, numNeigh, idxNeigh, ifree, ibase, idxBase, idxDiff)
    for (iatom = 0; iatom < lenAtoms; ++iatom)
    {
        for (ibase = 0; ibase < this->numBasis; ++ibase)
        {
            symmData[iatom][ibase] = ZERO;
        }

        for (ibase = 0; ibase < this->numBasis; ++ibase)
        {
            symmDiff[iatom][ibase + 0 * this->numBasis] = ZERO;
            symmDiff[iatom][ibase + 1 * this->numBasis] = ZERO;
            symmDiff[iatom][ibase + 2 * this->numBasis] = ZERO;
        }

        numNeigh = this->numNeighs[iatom];
        idxNeigh = this->idxNeighs[iatom];

        if (numNeigh < 1 || idxNeigh < 0)
        {
            continue;
        }

        for (ineigh = 0; ineigh < numNeigh; ++ineigh)
        {
            ifree   = 3 * (ineigh + 1);
            jneigh  = ineigh + idxNeigh;
            idxBase = jneigh * this->numBasis;

            for (ibase = 0; ibase < this->numBasis; ++ibase)
            {
                symmData[iatom][ibase] += this->symmDataAll[ibase + idxBase];
            }

            for (ibase = 0; ibase < this->numBasis; ++ibase)
            {
                idxDiff = (ibase + idxBase) * 3;
                symmDiff[iatom][ibase + 0 * this->numBasis] -= this->symmDiffAll[idxDiff + 0];
                symmDiff[iatom][ibase + 1 * this->numBasis] -= this->symmDiffAll[idxDiff + 1];
                symmDiff[iatom][ibase + 2 * this->numBasis] -= this->symmDiffAll[idxDiff + 2];
                symmDiff[iatom][ibase + (ifree + 0) * this->numBasis] = this->symmDiffAll[idxDiff + 0];
                symmDiff[iatom][ibase + (ifree + 1) * this->numBasis] = this->symmDiffAll[idxDiff + 1];
                symmDiff[iatom][ibase + (ifree + 2) * this->numBasis] = this->symmDiffAll[idxDiff + 2];
            }
        }
    }

    // check error of cuda
    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        char message[512];
        sprintf(message, "hipError_t: %s\n", hipGetErrorString(error));
        stop_by_error(message);
    }
}
