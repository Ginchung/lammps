#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 AdvanceSoft Corporation
 *
 * This software is released under the MIT License.
 * http://opensource.org/licenses/mit-license.php
 */

#include "nnp_symm_func_gpu.h"

SymmFuncGPU::SymmFuncGPU(int numElems, bool tanhCutFunc, bool elemWeight, int sizeRad, int sizeAng,
                         nnpreal rcutRad, nnpreal rcutAng, int cutoffMode) : SymmFunc(numElems, tanhCutFunc, elemWeight)
{
    if (sizeRad < 1)
    {
        stop_by_error("size of radius basis is not positive.");
    }

    if (sizeAng < 0)
    {
        stop_by_error("size of angle basis is negative.");
    }

    if (rcutRad <= ZERO)
    {
        stop_by_error("cutoff for radius is not positive.");
    }

    if (sizeAng > 0 && rcutAng <= ZERO)
    {
        stop_by_error("cutoff for angle is not positive.");
    }

    this->transDiff  = true;

#ifdef SYMMDIFF_HIDDEN
    this->hiddenDiff = true;
#else
    this->hiddenDiff = false;
#endif

    this->maxThreadsPerBlock = 1;

    this->sizeRad = sizeRad;
    this->sizeAng = sizeAng;

    if (this->elemWeight)
    {
        this->numRadBasis = this->sizeRad;
        this->numAngBasis = this->sizeAng;
    }
    else
    {
        this->numRadBasis = this->sizeRad * this->numElems;
        this->numAngBasis = this->sizeAng * (this->numElems * (this->numElems + 1) / 2);
    }

    this->numBasis = this->numRadBasis + this->numAngBasis;

    this->rcutRad = rcutRad;
    this->rcutAng = rcutAng;

    this->sizeLenAtoms  = 0;
    this->sizeMaxAtoms  = 0;
    this->sizeTotNeigh1 = 0;
    this->sizeTotNeigh2 = 0;
    this->sizeFullNeigh = 0;

    if (cutoffMode == CUTOFF_MODE_SINGLE)
    {
        this->sizePosNeighbor = 6;
    }
    else if (cutoffMode == CUTOFF_MODE_DOUBLE || cutoffMode == CUTOFF_MODE_IPSO)
    {
        this->sizePosNeighbor = 8;
    }
    else
    {
        this->sizePosNeighbor = 4;
    }

    this->numNeighs        = nullptr;
    this->numNeighs_d      = nullptr;
    this->idxNeighs        = nullptr;
    this->idxNeighs_d      = nullptr;
    this->elementAll       = nullptr;
    this->elementAll_d     = nullptr;
    this->posNeighborAll   = nullptr;
    this->posNeighborAll_d = nullptr;
    this->symmDataSum      = nullptr;
    this->symmDataSum_d    = nullptr;
    this->symmDataAll_d    = nullptr;
    this->symmDiffAll      = nullptr;
    this->symmDiffAll_d    = nullptr;
    this->symmDiffFull_d   = nullptr;
    this->symmGrad         = nullptr;
    this->symmGrad_d       = nullptr;
    this->forceData        = nullptr;
    this->forceData_d      = nullptr;

#ifdef _NNP_SINGLE
    hipError_t error = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
#else
    hipError_t error = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
#endif

    if (error != hipSuccess)
    {
        char message[512];
        sprintf(message, "error of hipDeviceSetSharedMemConfig: %s\n", hipGetErrorString(error));
        stop_by_error(message);
    }
}

SymmFuncGPU::~SymmFuncGPU()
{
    if (this->numNeighs        != nullptr) hipHostFree(this->numNeighs);
    if (this->numNeighs_d      != nullptr) hipFree    (this->numNeighs_d);
    if (this->idxNeighs        != nullptr) hipHostFree(this->idxNeighs);
    if (this->idxNeighs_d      != nullptr) hipFree    (this->idxNeighs_d);
    if (this->elementAll       != nullptr) hipHostFree(this->elementAll);
    if (this->elementAll_d     != nullptr) hipFree    (this->elementAll_d);
    if (this->posNeighborAll   != nullptr) hipHostFree(this->posNeighborAll);
    if (this->posNeighborAll_d != nullptr) hipFree    (this->posNeighborAll_d);
    if (this->symmDataSum      != nullptr) hipHostFree(this->symmDataSum);
    if (this->symmDataSum_d    != nullptr) hipFree    (this->symmDataSum_d);
    if (this->symmDataAll_d    != nullptr) hipFree    (this->symmDataAll_d);
    if (this->symmDiffAll      != nullptr) hipHostFree(this->symmDiffAll);
    if (this->symmDiffAll_d    != nullptr) hipFree    (this->symmDiffAll_d);
    if (this->symmDiffFull_d   != nullptr) hipFree    (this->symmDiffFull_d);
    if (this->symmGrad         != nullptr) hipHostFree(this->symmGrad);
    if (this->symmGrad_d       != nullptr) hipFree    (this->symmGrad);
    if (this->forceData        != nullptr) hipHostFree(this->forceData);
    if (this->forceData_d      != nullptr) hipFree    (this->forceData_d);
}

__global__ void sumupSymmData(int* numNeighs, int* idxNeighs, nnpreal* symmData, nnpreal* symmDataSum)
{
    const int iatom    = blockIdx.x;
    const int ibase    = threadIdx.x;
    const int numBasis = blockDim.x;
    const int numNeigh = numNeighs[iatom];
    const int idxNeigh = idxNeighs[iatom];
    const int idxData  = ibase * numNeigh + numBasis * idxNeigh;

    int ineigh;

    nnpreal symmData0 = ZERO;

    for (ineigh = 0; ineigh < numNeigh; ++ineigh)
    {
        symmData0 += symmData[ineigh + idxData];
    }

    symmDataSum[ibase + iatom * numBasis] = symmData0;
}

__global__ void multSymmDiff(int* numNeighs, int* idxNeighs,
                             nnpreal* symmDiff, nnpreal* symmGrad, nnpreal** forceData, int numBasis)
{
    const int iatom    = blockIdx.x;
    const int ineigh   = threadIdx.x;
    const int numNeigh = numNeighs[iatom];
    const int idxNeigh = idxNeighs[iatom];
    const int idxData0 = ineigh + numBasis * idxNeigh;
    const int idxBasis = iatom * numBasis;
    const int idxForce = (ineigh + idxNeigh) * 3;

    if (ineigh >= numNeigh)
    {
        return;
    }

    int ibase;
    int idxData;
    int idxDiff;

    nnpreal symmGrad0;

    nnpreal forceX = ZERO;
    nnpreal forceY = ZERO;
    nnpreal forceZ = ZERO;

    for (ibase = 0; ibase < numBasis; ++ibase)
    {
        idxData = ibase * numNeigh + idxData0;
        idxDiff = idxData * 3;

        symmGrad0 = symmGrad[ibase + idxBasis];

        forceX += symmDiff[idxDiff + 0] * symmGrad0;
        forceY += symmDiff[idxDiff + 1] * symmGrad0;
        forceZ += symmDiff[idxDiff + 2] * symmGrad0;
    }

    forceData[idxForce + 0] = forceX;
    forceData[idxForce + 1] = forceY;
    forceData[idxForce + 2] = forceZ;
}

void SymmFuncGPU::calculate(int lenAtoms, int* numNeighbor, int* idxNeighbor, int** elemNeighbor, nnpreal*** posNeighbor,
                            nnpreal* symmData, nnpreal* symmDiff)
{
    if (lenAtoms < 0)
    {
        stop_by_error("#atoms is not positive.");
    }

    if (numNeighbor == nullptr || idxNeighbor == nullptr || elemNeighbor == nullptr || posNeighbor == nullptr)
    {
        stop_by_error("neighbor is null.");
    }

    if (symmData == nullptr)
    {
        stop_by_error("symmData is null.");
    }

#ifndef SYMMDIFF_HIDDEN
    if (symmDiff == nullptr)
    {
        stop_by_error("symmDiff is null.");
    }
#endif

    if (this->numBasis > this->maxThreadsPerBlock)
    {
        stop_by_error("too less #threads a block for GPU (#threads < numBasis).");
    }

    // define varialbes
    int iatom;
    int ineigh, jneigh;
    int idata;
    int ipos;

    int numNeigh;
    int idxNeigh;
    int maxNeigh;
    int totNeigh;

    int numData;

    int numPos;
    int idxPos;

    int numModeBatchs;
    int modesPerBatch;
    int dimBasis;

    dim3 grid;
    dim3 block;

    size_t sizeShared;

    // allocate memory about lenAtoms
    if (this->sizeLenAtoms < lenAtoms)
    {
        if (this->numNeighs   != nullptr) hipHostFree(this->numNeighs);
        if (this->numNeighs_d != nullptr) hipFree    (this->numNeighs_d);
        if (this->idxNeighs   != nullptr) hipHostFree(this->idxNeighs);
        if (this->idxNeighs_d != nullptr) hipFree    (this->idxNeighs_d);

        hipHostMalloc(&(this->numNeighs),   sizeof(int) * lenAtoms);
        hipMalloc    (&(this->numNeighs_d), sizeof(int) * lenAtoms);
        hipHostMalloc(&(this->idxNeighs),   sizeof(int) * lenAtoms);
        hipMalloc    (&(this->idxNeighs_d), sizeof(int) * lenAtoms);

        this->sizeLenAtoms = lenAtoms;
    }

    // count neighbors
    maxNeigh = 0;
    totNeigh = numNeighbor[lenAtoms - 1] + idxNeighbor[lenAtoms - 1] - idxNeighbor[0];

    #pragma omp parallel for private(iatom, numNeigh, idxNeigh) reduction(max:maxNeigh)
    for (iatom = 0; iatom < lenAtoms; ++iatom)
    {
        numNeigh = numNeighbor[iatom];
        idxNeigh = idxNeighbor[iatom] - idxNeighbor[0];
        this->numNeighs[iatom] = numNeigh;
        this->idxNeighs[iatom] = idxNeigh;

        maxNeigh  = max(maxNeigh, numNeigh);
    }

    if (maxNeigh > this->maxThreadsPerBlock)
    {
        stop_by_error("too less #threads a block for GPU (#threads < maxNeigh).");
    }

    if (maxNeigh < 1 || totNeigh < 1)
    {
        // because this->numNeighs[iatom] is always 0,
        // there is no need to do symmDiff = ZERO.

        numData = lenAtoms * this->numBasis;

        #pragma omp parallel for private (idata)
        for (idata = 0; idata < numData; ++idata)
        {
            symmData[idata] = ZERO;
        }

        return;
    }

    // allocate memory about totNeigh
    if (this->sizeTotNeigh1 < totNeigh)
    {
        if (this->elementAll       != nullptr) hipHostFree(this->elementAll);
        if (this->elementAll_d     != nullptr) hipFree    (this->elementAll_d);
        if (this->posNeighborAll   != nullptr) hipHostFree(this->posNeighborAll);
        if (this->posNeighborAll_d != nullptr) hipFree    (this->posNeighborAll_d);
        if (this->symmDataSum      != nullptr) hipHostFree(this->symmDataSum);
        if (this->symmDiffAll      != nullptr) hipHostFree(this->symmDiffAll);
        if (this->symmDataSum_d    != nullptr) hipFree    (this->symmDataSum_d);
        if (this->symmDataAll_d    != nullptr) hipFree    (this->symmDataAll_d);
        if (this->symmDiffAll_d    != nullptr) hipFree    (this->symmDiffAll_d);

        hipHostMalloc(&(this->elementAll),       sizeof(gint)    * totNeigh);
        hipMalloc    (&(this->elementAll_d),     sizeof(gint)    * totNeigh);
        hipHostMalloc(&(this->posNeighborAll),   sizeof(nnpreal) * this->sizePosNeighbor * totNeigh);
        hipMalloc    (&(this->posNeighborAll_d), sizeof(nnpreal) * this->sizePosNeighbor * totNeigh);
#ifndef SYMMFUNC_DIRECT_COPY
        hipHostMalloc(&(this->symmDataSum),      sizeof(nnpreal)     * lenAtoms * this->numBasis);
#ifndef SYMMDIFF_HIDDEN
        hipHostMalloc(&(this->symmDiffAll),      sizeof(nnpreal) * 3 * totNeigh * this->numBasis);
#endif
#endif
        hipMalloc    (&(this->symmDataSum_d),    sizeof(nnpreal)     * lenAtoms * this->numBasis);
        hipMalloc    (&(this->symmDataAll_d),    sizeof(nnpreal)     * totNeigh * this->numBasis);
#ifndef SYMMDIFF_HIDDEN
        hipMalloc    (&(this->symmDiffAll_d),    sizeof(nnpreal) * 3 * totNeigh * this->numBasis);
#endif

        this->sizeTotNeigh1 = totNeigh;
    }

    // serialize all data of neighbors
    #pragma omp parallel for private (iatom, ineigh, jneigh, numNeigh, idxNeigh, ipos, numPos, idxPos)
    for (iatom = 0; iatom < lenAtoms; ++iatom)
    {
        numNeigh = this->numNeighs[iatom];
        idxNeigh = this->idxNeighs[iatom];

        // element / atomnum of neighbor atoms
        for (ineigh = 0; ineigh < numNeigh; ++ineigh)
        {
            this->elementAll[ineigh + idxNeigh] = elemNeighbor[iatom][ineigh];
        }

        // positions of neighbor atoms
        for (ineigh = 0; ineigh < numNeigh; ++ineigh)
        {
            jneigh = ineigh + idxNeigh;
            numPos = this->sizePosNeighbor;
            idxPos = jneigh * numPos;

            for (ipos = 0; ipos < numPos; ++ipos)
            {
                this->posNeighborAll[ipos + idxPos] = posNeighbor[iatom][ineigh][ipos];
            }
        }
    }

    // copy memory host -> gpu
    hipMemcpy(this->numNeighs_d,      this->numNeighs,      sizeof(int)     * lenAtoms,                         hipMemcpyHostToDevice);
    hipMemcpy(this->idxNeighs_d,      this->idxNeighs,      sizeof(int)     * lenAtoms,                         hipMemcpyHostToDevice);
    hipMemcpy(this->elementAll_d,     this->elementAll,     sizeof(gint)    * totNeigh,                         hipMemcpyHostToDevice);
    hipMemcpy(this->posNeighborAll_d, this->posNeighborAll, sizeof(nnpreal) * totNeigh * this->sizePosNeighbor, hipMemcpyHostToDevice);

    /*
     * radial part
     *   imode   -> threadIdx.x + blockIdx.y * blockDim.x
     *   ineigh1 -> threadIdx.y
     *   iatom   -> blockIdx.x
     *   (num of mode batch) -> blockDim.x
     *   (idx of mode batch) -> blockIdx.y
     */
    if (this->sizeRad > 0)
    {
        this->getSizeOfModeBatchs(&numModeBatchs, &modesPerBatch, this->sizeRad, maxNeigh);

        block = dim3(modesPerBatch, maxNeigh, 1);
        grid  = dim3(lenAtoms, numModeBatchs, 1);

        this->calculateRadial(grid, block, idxNeighbor[0]);
    }

    /*
     * angular  part
     *   imode   -> threadIdx.x + blockIdx.y * blockDim.x
     *   ineigh1 -> threadIdx.y
     *   iatom   -> blockIdx.x
     *   (num of mode batch) -> blockDim.x
     *   (idx of mode batch) -> blockIdx.y
     */
    if (this->sizeAng > 0)
    {
        this->getSizeOfModeBatchs(&numModeBatchs, &modesPerBatch, this->sizeAng, maxNeigh);

        block = dim3(modesPerBatch, maxNeigh, 1);
        grid  = dim3(lenAtoms, numModeBatchs, 1);

        sizeShared = sizeof(gint)    * maxNeigh
                   + sizeof(nnpreal) * maxNeigh * 5;

        if (this->elemWeight)
        {
            this->calculateAnglarElemWeight(grid, block, sizeShared, idxNeighbor[0]);
        }
        else
        {
            dimBasis = this->numElems * (this->numElems + 1) / 2;

            if (dimBasis > MAX_ELEMENT_PAIRS)
            {
                stop_by_error("too much elements for symmetry functions on GPU, please use CPU.");
            }

            this->calculateAnglarNotElemWeight(grid, block, sizeShared, idxNeighbor[0], dimBasis);
        }
    }

    // sum up symmData
    block = dim3(this->numBasis, 1, 1);
    grid  = dim3(lenAtoms, 1, 1);

    sumupSymmData<<<grid, block>>>(
                 this->numNeighs_d, this->idxNeighs_d, this->symmDataAll_d, this->symmDataSum_d);

    // copy memory gpu -> host
#ifdef SYMMFUNC_DIRECT_COPY
    hipMemcpy(symmData, this->symmDataSum_d, sizeof(nnpreal)     * lenAtoms * this->numBasis, hipMemcpyDeviceToHost);
#ifndef SYMMDIFF_HIDDEN
    hipMemcpy(symmDiff, this->symmDiffAll_d, sizeof(nnpreal) * 3 * totNeigh * this->numBasis, hipMemcpyDeviceToHost);
#endif
#else
    hipMemcpy(this->symmDataSum, this->symmDataSum_d, sizeof(nnpreal)     * lenAtoms * this->numBasis, hipMemcpyDeviceToHost);
    memcpy    (      symmData,    this->symmDataSum,   sizeof(nnpreal)     * lenAtoms * this->numBasis);
#ifndef SYMMDIFF_HIDDEN
    hipMemcpy(this->symmDiffAll, this->symmDiffAll_d, sizeof(nnpreal) * 3 * totNeigh * this->numBasis, hipMemcpyDeviceToHost);
    memcpy    (      symmDiff,    this->symmDiffAll,   sizeof(nnpreal) * 3 * totNeigh * this->numBasis);
#endif
#endif

    // check error of cuda
    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        char message[512];
        sprintf(message, "hipError_t: %s\n", hipGetErrorString(error));
        stop_by_error(message);
    }
}

void SymmFuncGPU::allocHiddenDiff(int maxAtoms, int fullNeigh)
{
#ifdef SYMMDIFF_HIDDEN
    if (maxAtoms < 0)
    {
        stop_by_error("max of atoms is not positive.");
    }

    // allocate memory about maxAtoms
    if (this->sizeMaxAtoms < maxAtoms)
    {
        if (this->symmGrad   != nullptr) hipHostFree(this->symmGrad);
        if (this->symmGrad_d != nullptr) hipFree    (this->symmGrad_d);

        hipHostMalloc(&(this->symmGrad),   sizeof(int) * maxAtoms * this->numBasis);
        hipMalloc    (&(this->symmGrad_d), sizeof(int) * maxAtoms * this->numBasis);

        this->sizeMaxAtoms = maxAtoms;
    }

    if (fullNeigh < 1)
    {
        return;
    }

    // allocate memory about fullNeigh
    if (this->sizeFullNeigh < fullNeigh)
    {
        if (this->symmDiffFull_d != nullptr) hipFree(this->symmDiffFull_d);

        hipMalloc(&(this->symmDiffFull_d), sizeof(nnpreal) * 3 * fullNeigh * this->numBasis);

        this->sizeFullNeigh = fullNeigh;
    }
#endif
}

void SymmFuncGPU::driveHiddenDiff(int lenAtoms, int* numNeighbor, int* idxNeighbor, nnpreal* forceData)
{
#ifdef SYMMDIFF_HIDDEN
    if (lenAtoms < 0)
    {
        stop_by_error("#atoms is not positive.");
    }

    if (numNeighbor == nullptr || idxNeighbor == nullptr)
    {
        stop_by_error("neighbor is null.");
    }

    if (forceData == nullptr)
    {
        stop_by_error("forceData is null.");
    }

    // define varialbes
    int iatom;

    int numNeigh;
    int idxNeigh;
    int maxNeigh;
    int totNeigh;

    dim3 grid;
    dim3 block;

    // check sizeLenAtoms
    if (this->sizeLenAtoms < lenAtoms)
    {
        stop_by_error("size of atoms is not correct.");
    }

    // count neighbors
    maxNeigh = 0;
    totNeigh = numNeighbor[lenAtoms - 1] + idxNeighbor[lenAtoms - 1] - idxNeighbor[0];

    #pragma omp parallel for private(iatom, numNeigh, idxNeigh) reduction(max:maxNeigh)
    for (iatom = 0; iatom < lenAtoms; ++iatom)
    {
        numNeigh = numNeighbor[iatom];
        idxNeigh = idxNeighbor[iatom] - idxNeighbor[0];
        this->numNeighs[iatom] = numNeigh;
        this->idxNeighs[iatom] = idxNeigh;

        maxNeigh  = max(maxNeigh, numNeigh);
    }

    if (maxNeigh > this->maxThreadsPerBlock)
    {
        stop_by_error("too less #threads a block for GPU (#threads < maxNeigh).");
    }

    if (maxNeigh < 1 || totNeigh < 1)
    {
        // because this->numNeighs[iatom] is always 0,
        // there is no need to do forceData = ZERO.
        return;
    }

    // allocate memory about totNeigh
    if (this->sizeTotNeigh2 < totNeigh)
    {
        if (this->forceData   != nullptr) hipHostFree(this->forceData);
        if (this->forceData_d != nullptr) hipFree    (this->forceData_d);

#ifndef SYMMFUNC_DIRECT_COPY
        hipHostMalloc(&(this->forceData),   sizeof(nnpreal) * 3 * totNeigh);
#endif
        hipMalloc    (&(this->forceData_d), sizeof(nnpreal) * 3 * totNeigh);

        this->sizeTotNeigh2 = totNeigh;
    }

    // copy memory host -> gpu
    hipMemcpy(this->numNeighs_d, this->numNeighs, sizeof(int)     * lenAtoms,                  hipMemcpyHostToDevice);
    hipMemcpy(this->idxNeighs_d, this->idxNeighs, sizeof(int)     * lenAtoms,                  hipMemcpyHostToDevice);
    hipMemcpy(this->symmGrad_d,  this->symmGrad,  sizeof(nnpreal) * lenAtoms * this->numBasis, hipMemcpyHostToDevice);

    // forceData = symmDiff * symmGrad
    idxNeigh = idxNeighbor[0];
    idxDiff  = 3 * idxNeigh * this->numBasis;

    block = dim3(maxNeigh, 1, 1);
    grid  = dim3(lenAtoms, 1, 1);

    multSymmDiff<<<grid, block>>>(this->numNeighs_d, this->idxNeighs_d,
                                  &(this->symmDiffFull_d[idxDiff]), this->forceData_d, this->numBasis);

    // copy memory gpu -> host
#ifdef SYMMFUNC_DIRECT_COPY
    hipMemcpy(      forceData, this->forceData_d, sizeof(nnpreal) * 3 * totNeigh, hipMemcpyDeviceToHost);
#else
    hipMemcpy(this->forceData, this->forceData_d, sizeof(nnpreal) * 3 * totNeigh, hipMemcpyDeviceToHost);
    memcpy    (      forceData, this->forceData,   sizeof(nnpreal) * 3 * totNeigh);
#endif

    // check error of cuda
    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        char message[512];
        sprintf(message, "hipError_t: %s\n", hipGetErrorString(error));
        stop_by_error(message);
    }
#endif
}

